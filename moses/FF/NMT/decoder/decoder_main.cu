#include <cstdlib>
#include <iostream>
#include <string>
#include <algorithm>
#include <memory>
#include <boost/timer/timer.hpp>
#include <boost/program_options/options_description.hpp>
#include <boost/program_options/parsers.hpp>
#include <boost/program_options/variables_map.hpp>
#include <boost/lexical_cast.hpp>

#include "model.h"
#include "vocab.h"
#include "decoder/nmt_decoder.h"


void ProgramOptions(int argc, char *argv[],
    std::string& modelPath,
    std::string& svPath,
    std::string& tvPath,
    size_t& device) {
  bool help = false;

  namespace po = boost::program_options;
  po::options_description cmdline_options("Allowed options");
  cmdline_options.add_options()
    ("device,d", po::value(&device)->default_value(0),
     "CUDA Device")
    ("model,m", po::value(&modelPath)->required(),
     "Path to a model")
    ("source,s", po::value(&svPath)->required(),
     "Path to a source vocab file.")
    ("target,t", po::value(&tvPath)->required(),
     "Path to a target vocab file.")
    ("help,h", po::value(&help)->zero_tokens()->default_value(false),
     "Print this help message and exit.")
  ;
  po::variables_map vm;
  try {
    po::store(po::command_line_parser(argc, argv).
              options(cmdline_options).run(), vm);
    po::notify(vm);
  } catch (std::exception& e) {
    std::cout << "Error: " << e.what() << std::endl << std::endl;

    std::cout << "Usage: " + std::string(argv[0]) +  " [options]" << std::endl;
    std::cout << cmdline_options << std::endl;
    exit(0);
  }

  if (help) {
    std::cout << "Usage: " + std::string(argv[0]) +  " [options]" << std::endl;
    std::cout << cmdline_options << std::endl;
    exit(0);
  }
}

int main(int argc, char* argv[]) {
  std::string modelPath, srcVocabPath, trgVocabPath;
  size_t device = 0;
  ProgramOptions(argc, argv, modelPath, srcVocabPath, trgVocabPath, device);
  hipSetDevice(device);
  std::shared_ptr<Weights> model(new Weights(modelPath));
  std::shared_ptr<Vocab> srcVocab(new Vocab(srcVocabPath));
  std::shared_ptr<Vocab> trgVocab(new Vocab(trgVocabPath));

  NMTDecoder decoder(model, srcVocab, trgVocab);

  std::cerr << "Start translating...\n";

  std::ios_base::sync_with_stdio(false);

  std::string line;
  while(std::getline(std::cin, line)) {
    auto result = decoder.translate(line);
    for (auto& word: result.first) std::cout << word << " ";
    std::cout << std::endl;
  }
  return 0;
}
